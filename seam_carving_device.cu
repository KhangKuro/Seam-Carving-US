#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "./src/library.h"
using namespace std;

// Global variables
int WIDTH;  // Width variable
__device__ int d_WIDTH;  // Device-side width variable

// Sobel filter kernels
int xSobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
int ySobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};

// Constant memory for device
__constant__ int d_xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
__constant__ int d_ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

const int filterWidth = 3;  // Width of the filter

void checkInput(int argc, char **argv, int &width, int &height, uchar3 *&inPixels, int &targetedWidth, dim3 &blockSize) {
    // Check the number of arguments
    if (argc != 4 && argc != 6) {
        printf("The number of arguments is invalid\n");
        exit(EXIT_FAILURE);
    }

    // Read the file
    readPnm(argv[1], width, height, inPixels);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    WIDTH = width; // Assign the width
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &width, sizeof(int))); // Copy width to device constant

    // Check the width that the user wants to reach
    targetedWidth = atoi(argv[3]); // Convert user's input to integer

    // Validate user's chosen width
    if (targetedWidth <= 0 || targetedWidth >= width) {
        printf("Your chosen width must be between 0 and the current picture's width!\n");
        exit(EXIT_FAILURE);
    }

    // Handle block size
    if (argc == 6) {
        blockSize.x = atoi(argv[4]); // Set block x-size
        blockSize.y = atoi(argv[5]); // Set block y-size
    } 

    // Check if the GPU functions properly
    printDeviceInfo();
}

// HOST
int measurePixelEnergy(uint8_t *grayPixels, int row, int col, int width, int height) {
    int x_kernel = 0; // Initialize variable to store x-axis convolution result
    int y_kernel = 0; // Initialize variable to store y-axis convolution result

    for (int i = 0; i < 3; ++i) { // Loop through rows of the 3x3 filter matrix
        for (int j = 0; j < 3; ++j) { // Loop through columns of the 3x3 filter matrix

            // Ensure boundary conditions for the image
            // 0 <= row - 1 + i < height
            int r = min(max(0, row - 1 + i), height - 1); // Ensure row index stays within image boundaries
            // 0 <= col - 1 + j < width
            int c = min(max(0, col - 1 + j), width - 1); // Ensure column index stays within image boundaries

            uint8_t pixelVal = grayPixels[r * WIDTH + c]; // Access pixel value from the grayscale image

            // Apply Sobel filter convolution along x-axis and y-axis
            x_kernel += pixelVal * xSobel[i][j]; // Convolution with the x-axis Sobel filter
            y_kernel += pixelVal * ySobel[i][j]; // Convolution with the y-axis Sobel filter
        }
    }
    return abs(x_kernel) + abs(y_kernel); // Calculate energy by summing absolute values of the convolutions
}

void measureEnergyUps(int *energy, int *minEnergy, int width, int height) {
    // Copy the bottom row of energy to minEnergy
    int lastRowIdx = (height - 1) * width;
    for (int c = 0; c < width; ++c) {
        minEnergy[lastRowIdx + c] = energy[lastRowIdx + c];
    }

    // Start from the second last row and compute minEnergy upwards
    for (int r = height - 2; r >= 0; --r) {
        for (int c = 0; c < width; ++c) {
            int idx = r * WIDTH + c; // Current index in minEnergy
            int belowIdx = (r + 1) * WIDTH  + c; // Index of pixel directly below

            int min = minEnergy[belowIdx]; // Initialize minimum energy with the pixel below

            // Check energy values of neighboring pixels below and update minimum if necessary
            if (c > 0 && minEnergy[belowIdx - 1] < min) {
                min = minEnergy[belowIdx - 1];
            }
            if (c < width - 1 && minEnergy[belowIdx + 1] < min) {
                min = minEnergy[belowIdx + 1];
            }

            minEnergy[idx] = min + energy[idx]; // Update minEnergy for the current pixel
        }
    }
}

void colorizeEnergy(int *energy, uchar3 *colorPic, int width, int height) {
    int maxEnergy = 0; // Initialize maxEnergy

    // Find the maximum energy value
    for (int i = 0; i < width * height; ++i) {
        if (energy[i] > maxEnergy) {
            maxEnergy = energy[i];
        }
    }

    // Color the pixels based on normalized energy values
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int idx = y * width + x; // Calculate index for energy and color arrays

            // Normalize energy value to a range between 0 and 1
            float normalizedEnergy = (float)energy[idx] / maxEnergy;

            // Convert normalized energy to color values
            uchar3 color;
            color.x = (unsigned char)(125 * normalizedEnergy); // Red channel
            color.y = (unsigned char)(190 * normalizedEnergy); // Green channel
            color.z = (unsigned char)(190 * normalizedEnergy); // Blue channel

            // Assign the calculated color to the corresponding pixel in the output array
            colorPic[idx] = color;
        }
    }
}

void seamCarveHost(uchar3 *inPixels, int width, int height, int targetedWidth, uchar3 *outPixels, uchar3 *outPixelsColor) {
    GpuTimer timer;
    timer.Start();

    // Copy the input pixels to output pixels (initialization)
    memcpy(outPixels, inPixels, width * height * sizeof(uchar3));
    memcpy(outPixelsColor, inPixels, width * height * sizeof(uchar3));

    // Memory allocation for energy and minEnergy arrays
    int *energy = (int *)malloc(width * height * sizeof(int));
    int *minEnergy = (int *)malloc(width * height * sizeof(int));
    
    // Memory allocation and conversion of input RGB pixels to grayscale
    uint8_t *grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    convertRgb2Gray_host(inPixels, width, height, grayPixels);

    // Calculate energy for all pixels in the image
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            energy[r * WIDTH + c] = measurePixelEnergy(grayPixels, r, c, width, height);
        }
    }
    measureEnergyUps(energy, minEnergy, width, height);
    colorizeEnergy(minEnergy, outPixelsColor, width, height);

    while (width > targetedWidth) {
      // Calculate energy from the beginning. (go from top to bottom)
      measureEnergyUps(energy, minEnergy, width, height);

      // find min index of first row
      int minCol = 0, r = 0, prevMinCol;
      for (int c = 1; c < width; ++c) {
          if (minEnergy[r * WIDTH + c] < minEnergy[r * WIDTH + minCol])
              minCol = c;
      }

      // Find and remove seam from first to last row
      for (; r < height; ++r) {
          // remove seam pixel on row r
          for (int i = minCol; i < width - 1; ++i) {
              outPixels[r * WIDTH + i] = outPixels[r * WIDTH + i + 1];
              grayPixels[r * WIDTH + i] = grayPixels[r * WIDTH + i + 1];
              energy[r * WIDTH + i] = energy[r * WIDTH + i + 1];
          }
         
          // Update energy
          if (r > 0) {
              int affectedCol = max(0, prevMinCol - 2);

              while (affectedCol <= prevMinCol + 2 && affectedCol < width - 1) {
                  energy[(r - 1) * WIDTH + affectedCol] = measurePixelEnergy(grayPixels, r - 1, affectedCol, width - 1, height);
                  affectedCol += 1;
              }
          }

          // find to the bottom
          if (r < height - 1) {
              prevMinCol = minCol;

              int belowIdx = (r + 1) * WIDTH + minCol;
              int min = minEnergy[belowIdx], minColCpy = minCol;
              if (minColCpy > 0 && minEnergy[belowIdx - 1] < min) {
                  min = minEnergy[belowIdx - 1];
                  minCol = minColCpy - 1;
              }
              if (minColCpy < width - 1 && minEnergy[belowIdx + 1] < min) {
                  minCol = minColCpy + 1;
              }
          }
      }

      int affectedCol;
      for (affectedCol = max(0, minCol - 2); affectedCol <= minCol + 2 && affectedCol < width - 1; ++affectedCol) {
          energy[(height - 1) * WIDTH + affectedCol] = measurePixelEnergy(grayPixels, height - 1, affectedCol, width - 1, height);
      }

      --width;
    }


    // Free dynamically allocated memory
    free(grayPixels);
    free(minEnergy);
    free(energy);

    // Stop the timer and print the execution time for the host function
    timer.Stop();
    timer.printTime((char *)"host");
}


// Device
__global__ void kernelRgbToGray(uchar3 *inPixels, int width, int height, uint8_t *outPixels) {
    // Calculate the indices in the image for processing
    int r = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int c = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    if (r < height && c < width) { // Ensure threads are within the image bounds
        int i = r * width + c; // Calculate the linear index

        // Convert RGB pixel to grayscale using luminance formula
        outPixels[i] = 0.299f * inPixels[i].x + 0.587f * inPixels[i].y + 0.114f * inPixels[i].z;
    }
}

__global__ void calEnergy(uint8_t *inPixels, int width, int height, int *energy) {
    // Calculate the thread's row and column indices in the image
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the energy for each thread
    if (col < width && row < height) {
        int x_kernel = 0, y_kernel = 0;

        for (int i = 0; i < filterWidth; ++i) {
            for (int j = 0; j < filterWidth; ++j) {
                int readRow = min(max(row - (filterWidth >> 1) + i, 0), height - 1);
                int readCol = min(max(col - (filterWidth >> 1) + j, 0), width - 1);
                
                uint8_t closest = inPixels[readRow * d_WIDTH + readCol];
                int filterIdx = i * filterWidth + j;
                x_kernel += closest * d_xSobel[filterIdx]; // Convolution with x-axis Sobel filter
                y_kernel += closest * d_ySobel[filterIdx]; // Convolution with y-axis Sobel filter
            }
        }

        energy[row * d_WIDTH + col] = abs(x_kernel) + abs(y_kernel);
    }
}


__global__ void calEnergyUpsKernel(int *energy, int *minEnergy, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x >> 1; // Half the block size

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x; // Calculate column index

    if (fromRow == height - 1 && col < width) {
        minEnergy[fromRow * width + col] = energy[fromRow * width + col]; // Copy bottom row's energy to minEnergy
    }
    __syncthreads(); // Synchronize threads after copying bottom row

    // Interative computation of minimal energy upwards
    for (int stride = fromRow != height - 1 ? 0 : 1; stride < halfBlock && fromRow - stride >= 0; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow - stride;
            int curCol = col + stride;

            // Ensure within bounds and process only valid columns
            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_WIDTH + curCol;
                int belowIdx = (curRow + 1) * d_WIDTH + curCol;

                int min = minEnergy[belowIdx]; // Initialize minimum energy with the pixel below

                // Update minimum energy by considering neighboring pixels below
                if (curCol > 0 && minEnergy[belowIdx - 1] < min)
                    min = minEnergy[belowIdx - 1];
                
                if (curCol < width - 1 && minEnergy[belowIdx + 1] < min)
                    min = minEnergy[belowIdx + 1];
                
                minEnergy[idx] = min + energy[idx]; // Update minEnergy for the current pixel
            }
        }
        __syncthreads(); // Synchronize threads after updating minEnergy
    }
}


__global__ void energyToColorKernel(int *energy, uchar3 *colorPic, int width, int height, int maxEnergy) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the x index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate the y index

    if (x < width && y < height) { // Ensure thread is within image bounds
        int idx = y * width + x; // Compute linear index

        // Normalize energy value to a range between 0 and 1
        float normalizedEnergy = (float)energy[idx] / maxEnergy;

        // Convert normalized energy to color values (RGB)
        uchar3 color;
        color.x = (unsigned char)(125 * (normalizedEnergy));  // Red channel
        color.y = (unsigned char)(190 * (normalizedEnergy));  // Green channel
        color.z = (unsigned char)(190 * (normalizedEnergy));  // Blue channel

        colorPic[idx] = color; // Assign calculated color to corresponding pixel in output array
    }
}

__global__ void carvingKernel(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    int row = blockIdx.x;
    int leastSignificant = leastSignificantPixel[row];

    // Update pixels only for threads whose index is within the range of leastSignificant to width - 1
    for (int i = leastSignificant + threadIdx.x; i < width - 1; i += blockDim.x) {
        int baseIdx = row * d_WIDTH + i;
        outPixels[baseIdx] = outPixels[baseIdx + 1];
        grayPixels[baseIdx] = grayPixels[baseIdx + 1];
        energy[baseIdx] = energy[baseIdx + 1];
    }
}

void findSeam(int * minEnergy, int *leastSignificantPixel, int width, int height) {
    int minCol = 0, r = 0; 

    for (int c = 1; c < width; ++c)
        if (minEnergy[r * WIDTH + c] < minEnergy[r * WIDTH + minCol])
            minCol = c;
    
    for (; r < height; ++r) { 
        leastSignificantPixel[r] = minCol;
        if (r < height - 1) { 
            int belowIdx = (r + 1) * WIDTH + minCol;
            int min = minEnergy[belowIdx], minColCpy = minCol;

            if (minColCpy > 0 && minEnergy[belowIdx - 1] < min) {
                min = minEnergy[belowIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && minEnergy[belowIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}

__global__ void findSeamKernel(int *minEnergy, int *leastSignificantPixel, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 

    if (col >= width) return; 

    int minCol = 0, r = 0;

    if (row == 0) { 
        for (int c = 1; c < width; ++c) {
            if (minEnergy[r * d_WIDTH + c] < minEnergy[r * d_WIDTH + minCol]) {
                minCol = c;
            }
        }
    }

    __syncthreads(); 

    for (; r < height; ++r) {
        leastSignificantPixel[r] = minCol;

        if (r < height - 1) {
            int belowIdx = (r + 1) * d_WIDTH + minCol;
            int min = minEnergy[belowIdx], minColCpy = minCol;

            if (minColCpy > 0 && minEnergy[belowIdx - 1] < min) {
                min = minEnergy[belowIdx - 1];
                minCol = minColCpy - 1;
            }

            if (minColCpy < width - 1 && minEnergy[belowIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }

        __syncthreads(); 
    }
}


void seamCarveDevice(uchar3 *inPixels, int width, int height, int targetedWidth, uchar3 *outPixels, dim3 blockSize, uchar3 *outPixelsColor) {
    // GPU timer initialization
    GpuTimer timer;
    timer.Start();

    // Device memory allocation

    uchar3 *d_inPixels, *d_outPixels;
    uint8_t *d_grayPixels;
    int *d_energy, *d_minimalEnergy;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_outPixels, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));
    CHECK(hipMalloc(&d_minimalEnergy, width * height * sizeof(int)));

    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));

    // Host memory allocation
    int *energy = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int *minEnergy = (int *)malloc(width * height * sizeof(int));

	  int blockSizeDp = 256;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1) + 1;

    // Copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_outPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // Convert input image to grayscale on the device
    dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
    // convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    // hipDeviceSynchronize();
    // CHECK(hipGetLastError());
    uint8_t * grayPixels = (uint8_t *)malloc((width) * height * sizeof(uint8_t));
    uchar3 * tempPixels = (uchar3 *)malloc((width) * height * sizeof(uchar3));

    CHECK(hipMemcpy(tempPixels, d_inPixels, (width) * height * sizeof(uchar3), hipMemcpyDeviceToHost));
    
    convertRgb2Gray_host(tempPixels, width, height, grayPixels);

    CHECK(hipMemcpy(d_grayPixels, grayPixels, (width) * height * sizeof(uint8_t), hipMemcpyHostToDevice));

    free(tempPixels);
    free(grayPixels);

    // update energy
    calEnergy<<<gridSize, blockSize>>>(d_grayPixels, width, height, d_energy);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    // Compute minimal seam table upwards in parallel
    for (int i = height - 1; i >= 0; i -= (stripHeight >> 1)) {
        calEnergyUpsKernel<<<gridSizeDp, blockSizeDp>>>(d_energy, d_minimalEnergy, width, height, i);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
    }

    // Copy minimal energy table back to host memory to find max energy
    CHECK(hipMemcpy(minEnergy, d_minimalEnergy, WIDTH * height * sizeof(int), hipMemcpyDeviceToHost));
    int maxEnergy = 0;
    for (int i = 0; i < width * height; ++i) {
        if (minEnergy[i] > maxEnergy) {
            maxEnergy = minEnergy[i];
        }
    }
    // Convert energy values to color representation
    energyToColorKernel<<<gridSize, blockSize>>>(d_minimalEnergy, d_outPixels, width, height, maxEnergy);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());


    while (width > targetedWidth) {
        // update energy
        calEnergy<<<gridSize, blockSize>>>(d_grayPixels, width, height, d_energy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // Compute minimal seam table upwards in parallel
        for (int i = height - 1; i >= 0; i -= (stripHeight >> 1)) {
        calEnergyUpsKernel<<<gridSizeDp, blockSizeDp>>>(d_energy, d_minimalEnergy, width, height, i);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        }

        int numThreadsPerBlock = 256;
        int numBlocks = (width + numThreadsPerBlock - 1) / numThreadsPerBlock;
        findSeamKernel<<<numBlocks, numThreadsPerBlock>>>(d_minimalEnergy, d_leastSignificantPixel, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_energy, width);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        
        --width;
    }

    // Copy processed pixels back to host memory
    CHECK(hipMemcpy(outPixels, d_inPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(outPixelsColor, d_outPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost));
    

    // Free device memory
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_outPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_minimalEnergy));

    // Free host memory
    free(minEnergy);
    free(leastSignificantPixel);
    free(energy);

    // Stop timer and print the execution time for the device function
    timer.Stop();
    timer.printTime((char *)"device");
}

// Main
int main(int argc, char **argv) {
    int width, height, targetedWidth;
    uchar3 *inPixels;
    dim3 blockSize(32, 32);

    // Check user's input
    checkInput(argc, argv, width, height, inPixels, targetedWidth, blockSize);

    // HOST: Perform energy calculation and color transformation on the CPU (host)
    uchar3 *out_host = (uchar3 *)malloc(width * height * sizeof(uchar3));
    uchar3 *out_host_color = (uchar3 *)malloc(width * height * sizeof(uchar3));
    seamCarveHost(inPixels, width, height, targetedWidth, out_host, out_host_color);

    // DEVICE: Perform energy calculation and color transformation on the GPU (device)
    uchar3 *out_device = (uchar3 *)malloc(width * height * sizeof(uchar3));
    uchar3 *out_device_color = (uchar3 *)malloc(width * height * sizeof(uchar3));
    seamCarveDevice(inPixels, width, height, targetedWidth, out_device, blockSize, out_device_color);

    // Compute error between device and host results
    printError((char *)"Error between device color result and host color result: ", out_host_color, out_device_color, targetedWidth, height);
    printError((char *)"Error between device result and host result: ", out_host, out_device, targetedWidth, height);

    // Write results to files
    printf("\nImage color energy output size (width x height) host: %i x %i\n", width, height);
    writePnm(out_host_color, width, height, width, concatStr(argv[2], "_energy_host.pnm"));

    printf("\nImage output size (width x height) host: %i x %i\n", targetedWidth, height);
    writePnm(out_host, targetedWidth, height, width, concatStr(argv[2], "_host.pnm"));

    printf("\nImage color energy output size (width x height) device: %i x %i\n", width, height);
    writePnm(out_device_color, width, height, width, concatStr(argv[2], "_energy_device.pnm"));

    printf("\nImage output size (width x height) device: %i x %i\n", targetedWidth, height);
    writePnm(out_device, targetedWidth, height, width, concatStr(argv[2], "_device.pnm"));

    // Free allocated memory
    free(inPixels);
    free(out_host);
    free(out_device);
}

