#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "./src/library.h"
using namespace std;

// Global variables
int WIDTH;  // Width variable
__device__ int d_WIDTH;  // Device-side width variable

// Sobel filter kernels
int xSobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
int ySobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};

// Constant memory for device
__constant__ int d_xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
__constant__ int d_ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

const int filterWidth = 3;  // Width of the filter

void checkInput(int argc, char **argv, int &width, int &height, uchar3 *&inPixels, int &desiredWidth, dim3 &blockSize) {
    // Checking the number of arguments
    if (argc != 4 && argc != 6) {
        printf("The number of arguments is invalid\n");
        exit(EXIT_FAILURE);
    }

    // Read file
    readPnm(argv[1], width, height, inPixels);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    WIDTH = width; // Assigning width
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &width, sizeof(int))); // Copy width to device constant

    // Check user's desired width
    desiredWidth = atoi(argv[3]); // Convert user input to integer

    // Validate user's desired width
    if (desiredWidth <= 0 || desiredWidth >= width) {
        printf("Your desired width must be between 0 and the current picture's width!\n");
        exit(EXIT_FAILURE);
    }

    // Block size handling
    if (argc == 6) {
        blockSize.x = atoi(argv[4]); // Set block x-size
        blockSize.y = atoi(argv[5]); // Set block y-size
    } 

    // Checking if the GPU is functioning properly
    printDeviceInfo();
}

// HOST
int getPixelEnergy(uint8_t *grayPixels, int row, int col, int width, int height) {
    int x_kernel = 0; // Initialize variable to store x-axis convolution result
    int y_kernel = 0; // Initialize variable to store y-axis convolution result

    for (int i = 0; i < 3; ++i) { // Loop through rows of the 3x3 filter matrix
        for (int j = 0; j < 3; ++j) { // Loop through columns of the 3x3 filter matrix

            // Ensure boundary conditions for the image
            // 0 <= row - 1 + i < height
            int r = min(max(0, row - 1 + i), height - 1); // Ensure row index stays within image boundaries
            // 0 <= col - 1 + j < width
            int c = min(max(0, col - 1 + j), width - 1); // Ensure column index stays within image boundaries

            uint8_t pixelVal = grayPixels[r * WIDTH + c]; // Access pixel value from the grayscale image

            // Apply Sobel filter convolution along x-axis and y-axis
            x_kernel += pixelVal * xSobel[i][j]; // Convolution with the x-axis Sobel filter
            y_kernel += pixelVal * ySobel[i][j]; // Convolution with the y-axis Sobel filter
        }
    }
    return abs(x_kernel) + abs(y_kernel); // Calculate energy by summing absolute values of the convolutions
}

void calculateEnergyUpwards(int *energy, int *minimalEnergy, int width, int height) {
    // Copy the bottom row of energy to minimalEnergy
    int lastRowIdx = (height - 1) * width;
    for (int c = 0; c < width; ++c) {
        minimalEnergy[lastRowIdx + c] = energy[lastRowIdx + c];
    }

    // Start from the second last row and compute minimalEnergy upwards
    for (int r = height - 2; r >= 0; --r) {
        for (int c = 0; c < width; ++c) {
            int idx = r * WIDTH + c; // Current index in minimalEnergy
            int belowIdx = (r + 1) * WIDTH  + c; // Index of pixel directly below

            int min = minimalEnergy[belowIdx]; // Initialize minimum energy with the pixel below

            // Check energy values of neighboring pixels below and update minimum if necessary
            if (c > 0 && minimalEnergy[belowIdx - 1] < min) {
                min = minimalEnergy[belowIdx - 1];
            }
            if (c < width - 1 && minimalEnergy[belowIdx + 1] < min) {
                min = minimalEnergy[belowIdx + 1];
            }

            minimalEnergy[idx] = min + energy[idx]; // Update minimalEnergy for the current pixel
        }
    }
}

void energyToColor(int *energy, uchar3 *colorPic, int width, int height) {
    int maxEnergy = 0; // Initialize maxEnergy

    // Find the maximum energy value
    for (int i = 0; i < width * height; ++i) {
        if (energy[i] > maxEnergy) {
            maxEnergy = energy[i];
        }
    }

    // Color the pixels based on normalized energy values
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int idx = y * width + x; // Calculate index for energy and color arrays

            // Normalize energy value to a range between 0 and 1
            float normalizedEnergy = (float)energy[idx] / maxEnergy;

            // Convert normalized energy to color values
            uchar3 color;
            color.x = (unsigned char)(125 * normalizedEnergy); // Red channel
            color.y = (unsigned char)(190 * normalizedEnergy); // Green channel
            color.z = (unsigned char)(190 * normalizedEnergy); // Blue channel

            // Assign the calculated color to the corresponding pixel in the output array
            colorPic[idx] = color;
        }
    }
}

void hostSeamCarving(uchar3 *inPixels, int width, int height, int desiredWidth, uchar3 *outPixels, uchar3 *outPixelsColor) {
    GpuTimer timer;
    timer.Start();

    // Copy the input pixels to output pixels (initialization)
    memcpy(outPixels, inPixels, width * height * sizeof(uchar3));
    memcpy(outPixelsColor, inPixels, width * height * sizeof(uchar3));

    // Memory allocation for energy and minimalEnergy arrays
    int *energy = (int *)malloc(width * height * sizeof(int));
    int *minimalEnergy = (int *)malloc(width * height * sizeof(int));
    
    // Memory allocation and conversion of input RGB pixels to grayscale
    uint8_t *grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    convertRgb2Gray_host(inPixels, width, height, grayPixels);

    // Calculate energy for all pixels in the image
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            energy[r * WIDTH + c] = getPixelEnergy(grayPixels, r, c, width, height);
        }
    }
    calculateEnergyUpwards(energy, minimalEnergy, width, height);
    energyToColor(minimalEnergy, outPixelsColor, width, height);

    while (width > desiredWidth) {
      // Calculate energy from the beginning. (go from top to bottom)
      calculateEnergyUpwards(energy, minimalEnergy, width, height);

      // find min index of first row
      int minCol = 0, r = 0, prevMinCol;
      for (int c = 1; c < width; ++c) {
          if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
              minCol = c;
      }

      // Find and remove seam from first to last row
      for (; r < height; ++r) {
          // remove seam pixel on row r
          for (int i = minCol; i < width - 1; ++i) {
              outPixels[r * WIDTH + i] = outPixels[r * WIDTH + i + 1];
              grayPixels[r * WIDTH + i] = grayPixels[r * WIDTH + i + 1];
              energy[r * WIDTH + i] = energy[r * WIDTH + i + 1];
          }
          // outPixelsColor[r * WIDTH + minCol].x = 255; // Red channel
          // outPixelsColor[r * WIDTH + minCol].y = 0;   // Green channel
          // outPixelsColor[r * WIDTH + minCol].z = 0;   // Blue channel

          // Update energy
          if (r > 0) {
              int affectedCol = max(0, prevMinCol - 2);

              while (affectedCol <= prevMinCol + 2 && affectedCol < width - 1) {
                  energy[(r - 1) * WIDTH + affectedCol] = getPixelEnergy(grayPixels, r - 1, affectedCol, width - 1, height);
                  affectedCol += 1;
              }
          }

          // find to the bottom
          if (r < height - 1) {
              prevMinCol = minCol;

              int belowIdx = (r + 1) * WIDTH + minCol;
              int min = minimalEnergy[belowIdx], minColCpy = minCol;
              if (minColCpy > 0 && minimalEnergy[belowIdx - 1] < min) {
                  min = minimalEnergy[belowIdx - 1];
                  minCol = minColCpy - 1;
              }
              if (minColCpy < width - 1 && minimalEnergy[belowIdx + 1] < min) {
                  minCol = minColCpy + 1;
              }
          }
      }

      int affectedCol;
      for (affectedCol = max(0, minCol - 2); affectedCol <= minCol + 2 && affectedCol < width - 1; ++affectedCol) {
          energy[(height - 1) * WIDTH + affectedCol] = getPixelEnergy(grayPixels, height - 1, affectedCol, width - 1, height);
      }

      --width;
    }


    // Free dynamically allocated memory
    free(grayPixels);
    free(minimalEnergy);
    free(energy);

    // Stop the timer and print the execution time for the host function
    timer.Stop();
    timer.printTime((char *)"host");
}


// Device
__global__ void convertRgb2GrayKernel(uchar3 *inPixels, int width, int height, uint8_t *outPixels) {
    // Calculate the indices in the image for processing
    int r = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int c = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    if (r < height && c < width) { // Ensure threads are within the image bounds
        int i = r * width + c; // Calculate the linear index

        // Convert RGB pixel to grayscale using luminance formula
        outPixels[i] = 0.299f * inPixels[i].x + 0.587f * inPixels[i].y + 0.114f * inPixels[i].z;
    }
}

__global__ void calEnergy(uint8_t *inPixels, int width, int height, int *energy) {
    // Calculate the thread's row and column indices in the image
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Define the size of shared memory
    int s_width = blockDim.x + filterWidth - 1;
    int s_height = blockDim.y + filterWidth - 1;

    // Declare shared memory to load data from global memory
    extern __shared__ uint8_t s_inPixels[];

    // Calculate indices for loading data into shared memory in a tiled manner
    int readRow = row - (filterWidth >> 1), readCol, tmpRow, tmpCol;
    int firstReadCol = col - (filterWidth >> 1);
    int virtualRow, virtualCol;

    // Load data from global memory into shared memory
    for (virtualRow = threadIdx.y; virtualRow < s_height; readRow += blockDim.y, virtualRow += blockDim.y) {
        tmpRow = readRow;
        //0 <= readCol <= height-1
        readRow = min(max(readRow, 0), height - 1); // Boundary check for row index
        readCol = firstReadCol;
        virtualCol = threadIdx.x;

        for (; virtualCol < s_width; readCol += blockDim.x, virtualCol += blockDim.x) {
            tmpCol = readCol;
            // 0 <= readCol <= width-1
            readCol = min(max(readCol, 0), width - 1); // Boundary check for column index
            s_inPixels[virtualRow * s_width + virtualCol] = inPixels[readRow * d_WIDTH + readCol];
            readCol = tmpCol;
        }
        readRow = tmpRow;
    } 
    __syncthreads(); // Synchronize threads after data loading into shared memory

    // Each thread computes energy using the loaded data in shared memory
    int x_kernel = 0, y_kernel = 0;
    for (int i = 0; i < filterWidth; ++i) {
        for (int j = 0; j < filterWidth; ++j) {
            uint8_t closest = s_inPixels[(threadIdx.y + i) * s_width + threadIdx.x + j];
            int filterIdx = i * filterWidth + j;
            x_kernel += closest * d_xSobel[filterIdx]; // Convolution with x-axis Sobel filter
            y_kernel += closest * d_ySobel[filterIdx]; // Convolution with y-axis Sobel filter
        }
    }

    // Each thread writes the computed energy back to global memory
    if (col < width && row < height)
        energy[row * d_WIDTH + col] = abs(x_kernel) + abs(y_kernel);
}


__global__ void calculateEnergyUpwardsKernel(int *energy, int *minimalEnergy, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x >> 1; // Half the block size

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x; // Calculate column index

    if (fromRow == height - 1 && col < width) {
        minimalEnergy[fromRow * width + col] = energy[fromRow * width + col]; // Copy bottom row's energy to minimalEnergy
    }
    __syncthreads(); // Synchronize threads after copying bottom row

    // Iterative computation of minimal energy upwards
    for (int stride = fromRow != height - 1 ? 0 : 1; stride < halfBlock && fromRow - stride >= 0; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow - stride;
            int curCol = col + stride;

            // Ensure within bounds and process only valid columns
            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_WIDTH + curCol;
                int belowIdx = (curRow + 1) * d_WIDTH + curCol;

                int min = minimalEnergy[belowIdx]; // Initialize minimum energy with the pixel below

                // Update minimum energy by considering neighboring pixels below
                if (curCol > 0 && minimalEnergy[belowIdx - 1] < min)
                    min = minimalEnergy[belowIdx - 1];
                
                if (curCol < width - 1 && minimalEnergy[belowIdx + 1] < min)
                    min = minimalEnergy[belowIdx + 1];
                
                minimalEnergy[idx] = min + energy[idx]; // Update minimalEnergy for the current pixel
            }
        }
        __syncthreads(); // Synchronize threads after updating minimalEnergy
    }
}

__global__ void carvingKernel(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    int row = blockIdx.x;
    int baseIdx = row * d_WIDTH;
    for (int i = leastSignificantPixel[row]; i < width - 1; ++i) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        energy[baseIdx + i] = energy[baseIdx + i + 1];
    }
    
}

void findSeam(int * minimalEnergy, int *leastSignificantPixel, int width, int height) {
    int minCol = 0, r = 0; 

    for (int c = 1; c < width; ++c)
        if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
            minCol = c;
    
    for (; r < height; ++r) { 
        leastSignificantPixel[r] = minCol;
        if (r < height - 1) { 
            int belowIdx = (r + 1) * WIDTH + minCol;
            int min = minimalEnergy[belowIdx], minColCpy = minCol;

            if (minColCpy > 0 && minimalEnergy[belowIdx - 1] < min) {
                min = minimalEnergy[belowIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && minimalEnergy[belowIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}

__global__ void calEnergyAtPixels(uint8_t *grayPixels, int *energy, const int *leastSignificantPixel, int width, int height) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row >= height || col >= width) return;

    // Assuming energy calculation is based on the difference in pixel intensity
    if (col < width - 1) {
        int pixelIdx = row * width + col;
        energy[pixelIdx] = abs((int)grayPixels[pixelIdx] - (int)grayPixels[pixelIdx + 1]);
    }
}
void deviceSeamCarving(uchar3 *inPixels, int width, int height, int desiredWidth, uchar3 *outPixels, dim3 blockSize, uchar3 *outPixelsColor) {
    // GPU timer initialization
    GpuTimer timer;
    timer.Start();

    // Device memory allocation

    uchar3 *d_inPixels, *d_outPixels;
    uint8_t *d_grayPixels;
    int *d_energy, *d_minimalEnergy;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_outPixels, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));
    CHECK(hipMalloc(&d_minimalEnergy, width * height * sizeof(int)));

    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));

    // Host memory allocation
    int *energy = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int *minimalEnergy = (int *)malloc(width * height * sizeof(int));

    // Dynamic shared memory size for energy computation
    size_t smemSize = (blockSize.x + 2) * (blockSize.y + 2) * sizeof(uint8_t);
    
    // Calculate block and grid sizes for minimal energy computation
    int blockSizeDp = 1024;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1);


    // Convert input image to grayscale on the device
    dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

    uint8_t * grayPixels = (uint8_t *)malloc((width) * height * sizeof(uint8_t));
    
    convertRgb2Gray_host(inPixels, width, height, grayPixels);

    hipMemcpyAsync(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_grayPixels, grayPixels, (width) * height * sizeof(uint8_t), hipMemcpyHostToDevice);

    while (width > desiredWidth) {

        // update energy
        calEnergy<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, d_energy);
       
        // Compute minimal seam table upwards in parallel
        for (int i = height - 1; i >= 0; i -= (stripHeight >> 1)) {
            calculateEnergyUpwardsKernel<<<gridSizeDp, blockSizeDp>>>(d_energy, d_minimalEnergy, width, height, i);
        }

        // find least significant pixel index of each row and store in d_leastSignificantPixel (SEQUENTIAL, in kernel or host)
        CHECK(hipMemcpy(minimalEnergy, d_minimalEnergy, WIDTH * height * sizeof(int), hipMemcpyDeviceToHost));
        findSeam(minimalEnergy, leastSignificantPixel, width, height);

        // carve
        CHECK(hipMemcpy(d_leastSignificantPixel, leastSignificantPixel, height * sizeof(int), hipMemcpyHostToDevice));
        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_energy, width);
        --width;
    }

    // Copy processed pixels back to host memory
    hipMemcpyAsync(outPixels, d_inPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost);    

    // Free device memory
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_outPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_minimalEnergy));

    // Free host memory
    free(minimalEnergy);
    free(leastSignificantPixel);
    free(energy);
    free(grayPixels);

    // Stop timer and print the execution time for the device function
    timer.Stop();
    timer.printTime((char *)"device");
}

// Main
int main(int argc, char **argv) {
    int width, height, desiredWidth;
    uchar3 *inPixels;
    dim3 blockSize(32, 32);

    // Check user's input
    checkInput(argc, argv, width, height, inPixels, desiredWidth, blockSize);

    // HOST: Perform energy calculation and color transformation on the CPU (host)
    uchar3 *out_host = (uchar3 *)malloc(width * height * sizeof(uchar3));
    uchar3 *out_host_color = (uchar3 *)malloc(width * height * sizeof(uchar3));
    hostSeamCarving(inPixels, width, height, desiredWidth, out_host, out_host_color);

    // DEVICE: Perform energy calculation and color transformation on the GPU (device)
    uchar3 *out_device = (uchar3 *)malloc(width * height * sizeof(uchar3));
    uchar3 *out_device_color = (uchar3 *)malloc(width * height * sizeof(uchar3));
    deviceSeamCarving(inPixels, width, height, desiredWidth, out_device, blockSize, out_device_color);

    // Compute error between device and host results
    printError((char *)"Error between device result and host result: ", out_host, out_device, desiredWidth, height);

    // Write results to files
    printf("\nImage output size (width x height) host: %i x %i\n", desiredWidth, height);
    writePnm(out_host, desiredWidth, height, width, concatStr(argv[2], "_host.pnm"));
    printf("\nImage output size (width x height) device: %i x %i\n", desiredWidth, height);
    writePnm(out_device, desiredWidth, height, width, concatStr(argv[2], "_device.pnm"));

    // Free allocated memory
    free(inPixels);
    free(out_host);
    free(out_device);
}