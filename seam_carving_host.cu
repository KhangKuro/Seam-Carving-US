#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "./src/library.h"
using namespace std;

// Global variables
int WIDTH;  // Width variable

// Sobel filter kernels
int xSobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
int ySobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};


void checkInput(int argc, char **argv, int &width, int &height, uchar3 *&inPixels, int &targetedWidth, dim3 &blockSize) {
    // Check the number of arguments
    if (argc != 4 && argc != 6) {
        printf("The number of arguments is invalid\n");
        exit(EXIT_FAILURE);
    }

    // Read the file
    readPnm(argv[1], width, height, inPixels);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    WIDTH = width; // Assign the width
    // Check the width that the user wants to reach
    targetedWidth = atoi(argv[3]); // Convert user's input to integer

    // Validate user's chosen width
    if (targetedWidth <= 0 || targetedWidth >= width) {
        printf("Your chosen width must be between 0 and the current picture's width!\n");
        exit(EXIT_FAILURE);
    }

    // Handle block size
    if (argc == 6) {
        blockSize.x = atoi(argv[4]); // Set block x-size
        blockSize.y = atoi(argv[5]); // Set block y-size
    } 

    // Check if the GPU functions properly
    printDeviceInfo();
}

// HOST
int measurePixelEnergy(uint8_t *grayPixels, int row, int col, int width, int height) {
    int x_kernel = 0; // Initialize variable to store x-axis convolution result
    int y_kernel = 0; // Initialize variable to store y-axis convolution result

    for (int i = 0; i < 3; ++i) { // Loop through rows of the 3x3 filter matrix
        for (int j = 0; j < 3; ++j) { // Loop through columns of the 3x3 filter matrix

            // Ensure boundary conditions for the image
            // 0 <= row - 1 + i < height
            int r = min(max(0, row - 1 + i), height - 1); // Ensure row index stays within image boundaries
            // 0 <= col - 1 + j < width
            int c = min(max(0, col - 1 + j), width - 1); // Ensure column index stays within image boundaries

            uint8_t pixelVal = grayPixels[r * WIDTH + c]; // Access pixel value from the grayscale image

            // Apply Sobel filter convolution along x-axis and y-axis
            x_kernel += pixelVal * xSobel[i][j]; // Convolution with the x-axis Sobel filter
            y_kernel += pixelVal * ySobel[i][j]; // Convolution with the y-axis Sobel filter
        }
    }
    return abs(x_kernel) + abs(y_kernel); // Calculate energy by summing absolute values of the convolutions
}

void measureEnergyUps(int *energy, int *minimalEnergy, int width, int height) {
    // Copy the bottom row of energy to minimalEnergy
    int lastRowIdx = (height - 1) * width;
    for (int c = 0; c < width; ++c) {
        minimalEnergy[lastRowIdx + c] = energy[lastRowIdx + c];
    }

    // Start from the second last row and compute minimalEnergy upwards
    for (int r = height - 2; r >= 0; --r) {
        for (int c = 0; c < width; ++c) {
            int idx = r * WIDTH + c; // Current index in minimalEnergy
            int belowIdx = (r + 1) * WIDTH  + c; // Index of pixel directly below

            int min = minimalEnergy[belowIdx]; // Initialize minimum energy with the pixel below

            // Check energy values of neighboring pixels below and update minimum if necessary
            if (c > 0 && minimalEnergy[belowIdx - 1] < min) {
                min = minimalEnergy[belowIdx - 1];
            }
            if (c < width - 1 && minimalEnergy[belowIdx + 1] < min) {
                min = minimalEnergy[belowIdx + 1];
            }

            minimalEnergy[idx] = min + energy[idx]; // Update minimalEnergy for the current pixel
        }
    }
}

void colorizeEnergy(int *energy, uchar3 *colorPic, int width, int height) {
    int maxEnergy = 0; // Initialize maxEnergy

    // Find the maximum energy value
    for (int i = 0; i < width * height; ++i) {
        if (energy[i] > maxEnergy) {
            maxEnergy = energy[i];
        }
    }

    // Color the pixels based on normalized energy values
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int idx = y * width + x; // Calculate index for energy and color arrays

            // Normalize energy value to a range between 0 and 1
            float normalizedEnergy = (float)energy[idx] / maxEnergy;

            // Convert normalized energy to color values
            uchar3 color;
            color.x = (unsigned char)(125 * normalizedEnergy); // Red channel
            color.y = (unsigned char)(190 * normalizedEnergy); // Green channel
            color.z = (unsigned char)(190 * normalizedEnergy); // Blue channel

            // Assign the calculated color to the corresponding pixel in the output array
            colorPic[idx] = color;
        }
    }
}

void seamCarveHost(uchar3 *inPixels, int width, int height, int targetedWidth, uchar3 *outPixels, uchar3 *outPixelsColor) {
    GpuTimer timer;
    timer.Start();

    // Copy the input pixels to output pixels (initialization)
    memcpy(outPixels, inPixels, width * height * sizeof(uchar3));
    memcpy(outPixelsColor, inPixels, width * height * sizeof(uchar3));

    // Memory allocation for energy and minimalEnergy arrays
    int *energy = (int *)malloc(width * height * sizeof(int));
    int *minimalEnergy = (int *)malloc(width * height * sizeof(int));
    
    // Memory allocation and conversion of input RGB pixels to grayscale
    uint8_t *grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    convertRgb2Gray_host(inPixels, width, height, grayPixels);

    // Calculate energy for all pixels in the image
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            energy[r * WIDTH + c] = measurePixelEnergy(grayPixels, r, c, width, height);
        }
    }
    measureEnergyUps(energy, minimalEnergy, width, height);
    colorizeEnergy(minimalEnergy, outPixelsColor, width, height);

    while (width > targetedWidth) {
      // Calculate energy from the beginning. (go from top to bottom)
      measureEnergyUps(energy, minimalEnergy, width, height);

      // find min index of first row
      int minCol = 0, r = 0, prevMinCol;
      for (int c = 1; c < width; ++c) {
          if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
              minCol = c;
      }

      // Find and remove seam from first to last row
      for (; r < height; ++r) {
          // remove seam pixel on row r
          for (int i = minCol; i < width - 1; ++i) {
              outPixels[r * WIDTH + i] = outPixels[r * WIDTH + i + 1];
              grayPixels[r * WIDTH + i] = grayPixels[r * WIDTH + i + 1];
              energy[r * WIDTH + i] = energy[r * WIDTH + i + 1];
          }

          // Update energy
          if (r > 0) {
              int affectedCol = max(0, prevMinCol - 2);

              while (affectedCol <= prevMinCol + 2 && affectedCol < width - 1) {
                  energy[(r - 1) * WIDTH + affectedCol] = measurePixelEnergy(grayPixels, r - 1, affectedCol, width - 1, height);
                  affectedCol += 1;
              }
          }

          // find to the bottom
          if (r < height - 1) {
              prevMinCol = minCol;

              int belowIdx = (r + 1) * WIDTH + minCol;
              int min = minimalEnergy[belowIdx], minColCpy = minCol;
              if (minColCpy > 0 && minimalEnergy[belowIdx - 1] < min) {
                  min = minimalEnergy[belowIdx - 1];
                  minCol = minColCpy - 1;
              }
              if (minColCpy < width - 1 && minimalEnergy[belowIdx + 1] < min) {
                  minCol = minColCpy + 1;
              }
          }
      }

      int affectedCol;
      for (affectedCol = max(0, minCol - 2); affectedCol <= minCol + 2 && affectedCol < width - 1; ++affectedCol) {
          energy[(height - 1) * WIDTH + affectedCol] = measurePixelEnergy(grayPixels, height - 1, affectedCol, width - 1, height);
      }

      --width;
    }


    // Free dynamically allocated memory
    free(grayPixels);
    free(minimalEnergy);
    free(energy);

    // Stop the timer and print the execution time for the host function
    timer.Stop();
    timer.printTime((char *)"host");
}



// Main
int main(int argc, char **argv) {
    int width, height, targetedWidth;
    uchar3 *inPixels;
    dim3 blockSize(32, 32);

    // Check user's input
    checkInput(argc, argv, width, height, inPixels, targetedWidth, blockSize);

    // HOST: Perform energy calculation and color transformation on the CPU (host)
    uchar3 *out_host = (uchar3 *)malloc(width * height * sizeof(uchar3));
    uchar3 *out_host_color = (uchar3 *)malloc(width * height * sizeof(uchar3));
    seamCarveHost(inPixels, width, height, targetedWidth, out_host, out_host_color);

    // Write results to files
    printf("\nImage color energy output size (width x height): %i x %i\n", width, height);
    writePnm(out_host_color, width, height, width, concatStr(argv[2], "_energy_host.pnm"));

    printf("\nImage output size (width x height): %i x %i\n", targetedWidth, height);
    writePnm(out_host, targetedWidth, height, width, concatStr(argv[2], "_host.pnm"));

    // Free allocated memory
    free(inPixels);
    free(out_host);
}

